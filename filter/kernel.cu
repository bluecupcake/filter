#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "ColorsInversion.h"
#include "HorizontalFlip.h"
#include "Blur.h"

__global__ void ColorsInversion(unsigned char* Image, int Channels);
__global__ void HorizontalFlip(unsigned char* Image, int Width, int Channels);

__global__ void Blur(unsigned char* Input_Image, unsigned char* Output_Image, int imageWidth, int imageHeight, int channels, int blurRadius);


void inversion(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	ColorsInversion << <Grid_Image, 1 >> > (Dev_Input_Image, Channels);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

__global__ void Blur(unsigned char* Input_Image, unsigned char* Output_Image, int imageWidth, int imageHeight, int channels, int blurRadius)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < imageWidth && y < imageHeight)
    {
        float blurValue[4] = { 0.0f };
        int blurSize = 2 * blurRadius + 1;
        int blurArea = blurSize * blurSize;

        for (int c = 0; c < channels; ++c)
        {
            for (int i = -blurRadius; i <= blurRadius; ++i)
            {
                for (int j = -blurRadius; j <= blurRadius; ++j)
                {
                    int neighborX = x + j;
                    int neighborY = y + i;

                    // Handle boundary cases
                    if (neighborX < 0 || neighborX >= imageWidth || neighborY < 0 || neighborY >= imageHeight)
                    {
                        continue;
                    }

                    int offset = (neighborX + neighborY * imageWidth) * channels + c;
                    blurValue[c] += Input_Image[offset];
                }
            }

            blurValue[c] /= blurArea;
        }

        int outputOffset = (x + y * imageWidth) * channels;
        for (int c = 0; c < channels; ++c)
        {
            Output_Image[outputOffset + c] = blurValue[c];
        }
    }
}

void blur(unsigned char* Input_Image, int Height, int Width, int Channels, int blurRadius) {
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;

    hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);
    hipMalloc((void**)&Dev_Output_Image, Height * Width * Channels);

    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

    int blockSizeX = 16;
    int blockSizeY = 16;

    
    int gridDimX = (Width + blockSizeX - 1) / blockSizeX;
    int gridDimY = (Height + blockSizeY - 1) / blockSizeY;

    
    dim3 gridSize(gridDimX, gridDimY);
    dim3 blockSize(blockSizeX, blockSizeY);
    Blur << <gridSize, blockSize >> > (Dev_Input_Image, Dev_Output_Image, Width, Height, Channels, blurRadius);
    hipMemcpy(Input_Image, Dev_Output_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

    
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
}


void horizontalFlip(unsigned char* Input_Image, int Height, int Width, int Channels)
{
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in GPU
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels * sizeof(unsigned char));

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels * sizeof(unsigned char), hipMemcpyHostToDevice);

	dim3 blockSize(16, 16);
	dim3 Grid_Image((Width  + blockSize.x - 1)/ blockSize.x, (Height + blockSize.y - 1)/ blockSize.y);
	HorizontalFlip << <Grid_Image, blockSize >> > (Dev_Input_Image, Width, Channels);

	//copy processed data back to CPU from GPU
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

	//free GPU memory
	hipFree(Dev_Input_Image);
}



__global__ void ColorsInversion(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
	}
}

__global__ void HorizontalFlip(unsigned char* Image, int Width, int Channels)
{

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < Width / 2)
	{
		int frontIndex = (y * Width + x) * Channels;
		int backIndex = (y * Width + (Width - 1 - x)) * Channels;

		for (int c = 0; c < Channels; ++c)
		{
			unsigned char temp = Image[frontIndex + c];
			Image[frontIndex + c] = Image[backIndex + c];
			Image[backIndex + c] = temp;
		}
	}
}